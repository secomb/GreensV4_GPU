#include "hip/hip_runtime.h"
/***********************************************************
tissueGPU3.cu
GPU kernel to accumulate contributions of vessel source
strengths qv to tissue solute levels pt.
Each tissue point is assigned one thread.
TWS January 2012
************************************************************/
#include <stdio.h>
#include <cutil_inline.h>

__global__ void tissueGPU3Kernel(float *d_tissxyz, float *d_vessxyz, float *d_pt000, float *d_qv000,
	int nnt, int nnv, int is2d, float req, float r2d)
{
    int itp = blockDim.x * blockIdx.x + threadIdx.x;
	int jvp,nnv2=2*nnv;
	float p = 0., xt,yt,zt,x,y,z,dist2,gtv,req2=req*req,r2d2=r2d*r2d;
    if(itp < nnt){
		xt = d_tissxyz[itp];
		yt = d_tissxyz[itp+nnt];
		zt = d_tissxyz[itp+nnt*2];
		for(jvp=0; jvp<nnv; jvp++){
			x = d_vessxyz[jvp] - xt;
			y = d_vessxyz[jvp+nnv] - yt;
			z = d_vessxyz[jvp+nnv2] - zt;
			dist2 = x*x + y*y + z*z;
			if(dist2 < req2){
				if(is2d) gtv = log(r2d2/req2) + 1. - dist2/req2;
				else gtv = (1.5 - 0.5*dist2/req2)/req;
			}
			else{
				if(is2d) gtv = log(r2d2/dist2);
				else gtv = 1./sqrt(dist2);
			}
			p += d_qv000[jvp]*gtv;
		}
		d_pt000[itp] = p;
	}
}

extern "C" void tissueGPU3(float *d_tissxyz, float *d_vessxyz, float *d_pt000, float *d_qv000,
		int nnt, int nnv, int is2d, float req, float r2d)
{
	int threadsPerBlock = 256;
	int blocksPerGrid = (nnt + threadsPerBlock - 1) / threadsPerBlock;
	tissueGPU3Kernel<<<blocksPerGrid, threadsPerBlock>>>(d_tissxyz, d_vessxyz, d_pt000, d_qv000,
		nnt, nnv, is2d, req, r2d);
}